#include "hip/hip_runtime.h"
#ifndef RasterTransform
#define RasterTransform

#include "PrGPU/KernelSupport/KernelCore.h"  
#include "PrGPU/KernelSupport/KernelMemory.h"

#if GF_DEVICE_TARGET_DEVICE
#if GF_DEVICE_TARGET_HLSL
#define fmax max
#define fmin min
#define fmod(x,y) (x - y * floor(x/y))
#define fabsf abs
#define sinf sin
#define cosf cos
#define floorf floor
#define fmodf(x,y) fmod(x,y)
#endif

GF_DEVICE_FUNCTION float min_custom(float a, float b) {
    return a < b ? a : b;
}

GF_DEVICE_FUNCTION float max_custom(float a, float b) {
    return a > b ? a : b;
}

GF_DEVICE_FUNCTION float fmod_custom(float x, float y) {
    return x - y * floorf(x / y);
}

GF_KERNEL_FUNCTION(RasterTransformKernel,
    ((GF_PTR_READ_ONLY(float4))(inSrc))
    ((GF_PTR(float4))(outDst)),
    ((int)(inSrcPitch))
    ((int)(inDstPitch))
    ((int)(in16f))
    ((unsigned int)(inWidth))
    ((unsigned int)(inHeight))
    ((float)(inScale))
    ((float)(inAngle))
    ((float)(inOffsetX))
    ((float)(inOffsetY))
    ((int)(inMaskToLayer))
    ((float)(inAlpha))
    ((float)(inFill))
    ((int)(inSample))
    ((int)(inXTiles))
    ((int)(inYTiles))
    ((int)(inMirror)),
    ((uint2)(inXY)(KERNEL_XY)))
{
    if (inXY.x < inWidth && inXY.y < inHeight)
    {
        float width = (float)inWidth;
        float height = (float)inHeight;

        float st_x = (float)inXY.x / width;
        float st_y = (float)inXY.y / height;

        st_x -= inOffsetX / 500.0f;
        st_y -= inOffsetY / 500.0f;

        st_x -= 0.5f;
        st_y -= 0.5f;

        st_x *= width / height;

        float angle_rad = inAngle * 0.0174533f;
        float cos_angle = cosf(angle_rad);
        float sin_angle = sinf(angle_rad);

        float rotated_x = st_x * cos_angle - st_y * sin_angle;
        float rotated_y = st_x * sin_angle + st_y * cos_angle;
        st_x = rotated_x;
        st_y = rotated_y;

        st_x /= inScale;
        st_y /= inScale;

        st_x /= width / height;

        st_x += 0.5f;
        st_y += 0.5f;

        float sample_x = st_x * width;
        float sample_y = st_y * height;

        float4 transformedPixel = { 0.0f, 0.0f, 0.0f, 0.0f };
        float4 basePixel = { 0.0f, 0.0f, 0.0f, 0.0f };

        if (inMaskToLayer || inFill > 0.0001f) {
            basePixel = ReadFloat4(inSrc, inXY.y * inSrcPitch + inXY.x, !!in16f);
        }

        bool outsideBounds = false;

        if (inXTiles) {
            if (inMirror) {
                float fracPart = fmod_custom(fabsf(sample_x / width), 1.0f);
                int isOdd = (int)floorf(fabsf(sample_x / width)) & 1;
                sample_x = isOdd ? (1.0f - fracPart) * width : fracPart * width;
            }
            else {
                sample_x = fmod_custom(fmod_custom(sample_x, width) + width, width);
            }
        }
        else {
            if (sample_x < 0 || sample_x >= width) {
                outsideBounds = true;
            }
        }

        if (inYTiles) {
            if (inMirror) {
                float fracPart = fmod_custom(fabsf(sample_y / height), 1.0f);
                int isOdd = (int)floorf(fabsf(sample_y / height)) & 1;
                sample_y = isOdd ? (1.0f - fracPart) * height : fracPart * height;
            }
            else {
                sample_y = fmod_custom(fmod_custom(sample_y, height) + height, height);
            }
        }
        else {
            if (sample_y < 0 || sample_y >= height) {
                outsideBounds = true;
            }
        }

        if (!outsideBounds) {
            if (inSample == 0) {     
                unsigned int rounded_x = (unsigned int)(sample_x + 0.5f);
                unsigned int rounded_y = (unsigned int)(sample_y + 0.5f);

                rounded_x = min(rounded_x, inWidth - 1);
                rounded_y = min(rounded_y, inHeight - 1);

                transformedPixel = ReadFloat4(inSrc, rounded_y * inSrcPitch + rounded_x, !!in16f);
            }
            else {    
                unsigned int x1 = (unsigned int)sample_x;
                unsigned int y1 = (unsigned int)sample_y;
                unsigned int x2 = min(x1 + 1, inWidth - 1);
                unsigned int y2 = min(y1 + 1, inHeight - 1);

                float fx = sample_x - x1;
                float fy = sample_y - y1;

                float4 p11 = ReadFloat4(inSrc, y1 * inSrcPitch + x1, !!in16f);
                float4 p12 = ReadFloat4(inSrc, y1 * inSrcPitch + x2, !!in16f);
                float4 p21 = ReadFloat4(inSrc, y2 * inSrcPitch + x1, !!in16f);
                float4 p22 = ReadFloat4(inSrc, y2 * inSrcPitch + x2, !!in16f);

                transformedPixel.x = (1 - fx) * (1 - fy) * p11.x + fx * (1 - fy) * p12.x + (1 - fx) * fy * p21.x + fx * fy * p22.x;
                transformedPixel.y = (1 - fx) * (1 - fy) * p11.y + fx * (1 - fy) * p12.y + (1 - fx) * fy * p21.y + fx * fy * p22.y;
                transformedPixel.z = (1 - fx) * (1 - fy) * p11.z + fx * (1 - fy) * p12.z + (1 - fx) * fy * p21.z + fx * fy * p22.z;
                transformedPixel.w = (1 - fx) * (1 - fy) * p11.w + fx * (1 - fy) * p12.w + (1 - fx) * fy * p21.w + fx * fy * p22.w;
            }
        }

        float4 outPixel;

        if (inMaskToLayer) {
            float baseA = basePixel.w;
            float fillF = inFill;
            float alphaF = inAlpha;

            outPixel.x = basePixel.x * fillF * (1.0f - transformedPixel.w * baseA * alphaF) + transformedPixel.x * baseA * alphaF;
            outPixel.y = basePixel.y * fillF * (1.0f - transformedPixel.w * baseA * alphaF) + transformedPixel.y * baseA * alphaF;
            outPixel.z = basePixel.z * fillF * (1.0f - transformedPixel.w * baseA * alphaF) + transformedPixel.z * baseA * alphaF;
            outPixel.w = basePixel.w;
        }
        else if (inFill > 0.0001f) {
            float fillF = inFill;
            float alphaF = inAlpha;

            outPixel.x = basePixel.x * fillF * (1.0f - transformedPixel.w * alphaF) + transformedPixel.x * alphaF;
            outPixel.y = basePixel.y * fillF * (1.0f - transformedPixel.w * alphaF) + transformedPixel.y * alphaF;
            outPixel.z = basePixel.z * fillF * (1.0f - transformedPixel.w * alphaF) + transformedPixel.z * alphaF;
            outPixel.w = basePixel.w * fillF * (1.0f - transformedPixel.w * alphaF) + transformedPixel.w * alphaF;
        }
        else {
            float alphaF = inAlpha;
            outPixel.x = transformedPixel.x * alphaF;
            outPixel.y = transformedPixel.y * alphaF;
            outPixel.z = transformedPixel.z * alphaF;
            outPixel.w = transformedPixel.w * alphaF;
        }

        WriteFloat4(outPixel, outDst, inXY.y * inDstPitch + inXY.x, !!in16f);
    }
}
#endif
#if __NVCC__
void RasterTransform_CUDA(
    float const* src,
    float* dst,
    unsigned int srcPitch,
    unsigned int dstPitch,
    int is16f,
    unsigned int width,
    unsigned int height,
    float scale,
    float angle,
    float offsetX,
    float offsetY,
    int maskToLayer,
    float alpha,
    float fill,
    int sample,
    int xTiles,
    int yTiles,
    int mirror)
{
    dim3 blockDim(16, 16, 1);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y, 1);

    RasterTransformKernel << < gridDim, blockDim, 0 >> > (
        (float4 const*)src,
        (float4*)dst,
        srcPitch,
        dstPitch,
        is16f,
        width,
        height,
        scale,
        angle,
        offsetX,
        offsetY,
        maskToLayer,
        alpha,
        fill,
        sample,
        xTiles,
        yTiles,
        mirror
        );

    hipDeviceSynchronize();
}
#endif  
#endif