#include "hip/hip_runtime.h"
#ifndef AutoShake
#define AutoShake

#include "PrGPU/KernelSupport/KernelCore.h" 
#include "PrGPU/KernelSupport/KernelMemory.h"

#if GF_DEVICE_TARGET_DEVICE
#if GF_DEVICE_TARGET_HLSL
#define fmax max
#define fmin min
#define __device__
#define fabs abs
#define fmodf fmod
#define fmaxf max
#define fminf min


static const int p_array[256] = {
    151, 160, 137, 91, 90, 15, 131, 13, 201, 95, 96, 53, 194, 233, 7, 225,
    140, 36, 103, 30, 69, 142, 8, 99, 37, 240, 21, 10, 23, 190, 6, 148,
    247, 120, 234, 75, 0, 26, 197, 62, 94, 252, 219, 203, 117, 35, 11, 32,
    57, 177, 33, 88, 237, 149, 56, 87, 174, 20, 125, 136, 171, 168, 68, 175,
    74, 165, 71, 134, 139, 48, 27, 166, 77, 146, 158, 231, 83, 111, 229, 122,
    60, 211, 133, 230, 220, 105, 92, 41, 55, 46, 245, 40, 244, 102, 143, 54,
    65, 25, 63, 161, 1, 216, 80, 73, 209, 76, 132, 187, 208, 89, 18, 169,
    200, 196, 135, 130, 116, 188, 159, 86, 164, 100, 109, 198, 173, 186, 3, 64,
    52, 217, 226, 250, 124, 123, 5, 202, 38, 147, 118, 126, 255, 82, 85, 212,
    207, 206, 59, 227, 47, 16, 58, 17, 182, 189, 28, 42, 223, 183, 170, 213,
    119, 248, 152, 2, 44, 154, 163, 70, 221, 153, 101, 155, 167, 43, 172, 9,
    129, 22, 39, 253, 19, 98, 108, 110, 79, 113, 224, 232, 178, 185, 112, 104,
    218, 246, 97, 228, 251, 34, 242, 193, 238, 210, 144, 12, 191, 179, 162, 241,
    81, 51, 145, 235, 249, 14, 239, 107, 49, 192, 214, 31, 181, 199, 106, 157,
    184, 84, 204, 176, 115, 121, 50, 45, 127, 4, 150, 254, 138, 236, 205, 93,
    222, 114, 67, 29, 24, 72, 243, 141, 128, 195, 78, 66, 215, 61, 156, 180
};

static const float grad3_array[12][3] = {
    {1.0f, 1.0f, 0.0f}, {-1.0f, 1.0f, 0.0f}, {1.0f, -1.0f, 0.0f}, {-1.0f, -1.0f, 0.0f},
    {1.0f, 0.0f, 1.0f}, {-1.0f, 0.0f, 1.0f}, {1.0f, 0.0f, -1.0f}, {-1.0f, 0.0f, -1.0f},
    {0.0f, 1.0f, 1.0f}, {0.0f, -1.0f, 1.0f}, {0.0f, 1.0f, -1.0f}, {0.0f, -1.0f, -1.0f}
};

GF_DEVICE_FUNCTION int get_p(int idx) {
    return p_array[idx & 0xFF];
}

GF_DEVICE_FUNCTION void get_grad3(int idx, out float3 grad) {
    idx = idx % 12;
    grad.x = grad3_array[idx][0];
    grad.y = grad3_array[idx][1];
    grad.z = grad3_array[idx][2];
}
#else
__constant__ int p[256] = {
    151, 160, 137, 91, 90, 15, 131, 13, 201, 95, 96, 53, 194, 233, 7, 225,
    140, 36, 103, 30, 69, 142, 8, 99, 37, 240, 21, 10, 23, 190, 6, 148,
    247, 120, 234, 75, 0, 26, 197, 62, 94, 252, 219, 203, 117, 35, 11, 32,
    57, 177, 33, 88, 237, 149, 56, 87, 174, 20, 125, 136, 171, 168, 68, 175,
    74, 165, 71, 134, 139, 48, 27, 166, 77, 146, 158, 231, 83, 111, 229, 122,
    60, 211, 133, 230, 220, 105, 92, 41, 55, 46, 245, 40, 244, 102, 143, 54,
    65, 25, 63, 161, 1, 216, 80, 73, 209, 76, 132, 187, 208, 89, 18, 169,
    200, 196, 135, 130, 116, 188, 159, 86, 164, 100, 109, 198, 173, 186, 3, 64,
    52, 217, 226, 250, 124, 123, 5, 202, 38, 147, 118, 126, 255, 82, 85, 212,
    207, 206, 59, 227, 47, 16, 58, 17, 182, 189, 28, 42, 223, 183, 170, 213,
    119, 248, 152, 2, 44, 154, 163, 70, 221, 153, 101, 155, 167, 43, 172, 9,
    129, 22, 39, 253, 19, 98, 108, 110, 79, 113, 224, 232, 178, 185, 112, 104,
    218, 246, 97, 228, 251, 34, 242, 193, 238, 210, 144, 12, 191, 179, 162, 241,
    81, 51, 145, 235, 249, 14, 239, 107, 49, 192, 214, 31, 181, 199, 106, 157,
    184, 84, 204, 176, 115, 121, 50, 45, 127, 4, 150, 254, 138, 236, 205, 93,
    222, 114, 67, 29, 24, 72, 243, 141, 128, 195, 78, 66, 215, 61, 156, 180
};

__constant__ float grad3[12][3] = {
    {1.0f, 1.0f, 0.0f}, {-1.0f, 1.0f, 0.0f}, {1.0f, -1.0f, 0.0f}, {-1.0f, -1.0f, 0.0f},
    {1.0f, 0.0f, 1.0f}, {-1.0f, 0.0f, 1.0f}, {1.0f, 0.0f, -1.0f}, {-1.0f, 0.0f, -1.0f},
    {0.0f, 1.0f, 1.0f}, {0.0f, -1.0f, 1.0f}, {0.0f, 1.0f, -1.0f}, {0.0f, -1.0f, -1.0f}
};

GF_DEVICE_FUNCTION int get_p(int idx) {
    return p[idx & 0xFF];
}

GF_DEVICE_FUNCTION void get_grad3(int idx, float* grad) {
    idx = idx % 12;
    grad[0] = grad3[idx][0];
    grad[1] = grad3[idx][1];
    grad[2] = grad3[idx][2];
}
#endif

__device__ int min_int(int a, int b) {
    return a < b ? a : b;
}

__device__ unsigned int min_uint(unsigned int a, unsigned int b) {
    return a < b ? a : b;
}

__device__ int max_int(int a, int b) {
    return a > b ? a : b;
}

__device__ float fminf_custom(float a, float b) {
    return a < b ? a : b;
}

__device__ float fmaxf_custom(float a, float b) {
    return a > b ? a : b;
}

__device__ float fmodf_custom(float x, float y) {
    return x - y * floor(x / y);
}

#define F2_CONST 0.366025404f       
#define G2_CONST 0.211324865f       
#define F3_CONST 0.333333333f     
#define G3_CONST 0.166666667f     

GF_DEVICE_FUNCTION int fastfloor(float x) {
    int xi = (int)x;
    return x < xi ? xi - 1 : xi;
}

#if GF_DEVICE_TARGET_HLSL
GF_DEVICE_FUNCTION float dot_product(float3 g, float x, float y, float z) {
    return g.x * x + g.y * y + g.z * z;
}
#else
GF_DEVICE_FUNCTION float dot_product(float* g, float x, float y, float z) {
    return g[0] * x + g[1] * y + g[2] * z;
}
#endif

GF_DEVICE_FUNCTION float simplex_noise(float xin, float yin, float zin) {
    float n0, n1, n2, n3;       

    float s = (xin + yin + zin) * F3_CONST;         
    int i = fastfloor(xin + s);
    int j = fastfloor(yin + s);
    int k = fastfloor(zin + s);

    float t = (i + j + k) * G3_CONST;
    float X0 = i - t;         
    float Y0 = j - t;
    float Z0 = k - t;
    float x0 = xin - X0;        
    float y0 = yin - Y0;
    float z0 = zin - Z0;

    int i1, j1, k1;          
    int i2, j2, k2;          

    if (x0 >= y0) {
        if (y0 >= z0) {     
            i1 = 1; j1 = 0; k1 = 0; i2 = 1; j2 = 1; k2 = 0;
        }
        else if (x0 >= z0) {     
            i1 = 1; j1 = 0; k1 = 0; i2 = 1; j2 = 0; k2 = 1;
        }
        else {     
            i1 = 0; j1 = 0; k1 = 1; i2 = 1; j2 = 0; k2 = 1;
        }
    }
    else {  
        if (y0 < z0) {     
            i1 = 0; j1 = 0; k1 = 1; i2 = 0; j2 = 1; k2 = 1;
        }
        else if (x0 < z0) {     
            i1 = 0; j1 = 1; k1 = 0; i2 = 0; j2 = 1; k2 = 1;
        }
        else {     
            i1 = 0; j1 = 1; k1 = 0; i2 = 1; j2 = 1; k2 = 0;
        }
    }

    float x1 = x0 - i1 + G3_CONST;        
    float y1 = y0 - j1 + G3_CONST;
    float z1 = z0 - k1 + G3_CONST;
    float x2 = x0 - i2 + 2.0f * G3_CONST;        
    float y2 = y0 - j2 + 2.0f * G3_CONST;
    float z2 = z0 - k2 + 2.0f * G3_CONST;
    float x3 = x0 - 1.0f + 3.0f * G3_CONST;        
    float y3 = y0 - 1.0f + 3.0f * G3_CONST;
    float z3 = z0 - 1.0f + 3.0f * G3_CONST;

    int ii = i & 255;
    int jj = j & 255;
    int kk = k & 255;

    int gi0 = get_p(ii + get_p(jj + get_p(kk))) % 12;
    int gi1 = get_p(ii + i1 + get_p(jj + j1 + get_p(kk + k1))) % 12;
    int gi2 = get_p(ii + i2 + get_p(jj + j2 + get_p(kk + k2))) % 12;
    int gi3 = get_p(ii + 1 + get_p(jj + 1 + get_p(kk + 1))) % 12;

    float t0 = 0.5f - x0 * x0 - y0 * y0 - z0 * z0;
    if (t0 < 0) {
        n0 = 0.0f;
    }
    else {
        t0 *= t0;

#if GF_DEVICE_TARGET_HLSL
        float3 g0;
        get_grad3(gi0, g0);
        n0 = t0 * t0 * dot_product(g0, x0, y0, z0);
#else
        float g0[3];
        get_grad3(gi0, g0);
        n0 = t0 * t0 * dot_product(g0, x0, y0, z0);
#endif
    }

    float t1 = 0.5f - x1 * x1 - y1 * y1 - z1 * z1;
    if (t1 < 0) {
        n1 = 0.0f;
    }
    else {
        t1 *= t1;

#if GF_DEVICE_TARGET_HLSL
        float3 g1;
        get_grad3(gi1, g1);
        n1 = t1 * t1 * dot_product(g1, x1, y1, z1);
#else
        float g1[3];
        get_grad3(gi1, g1);
        n1 = t1 * t1 * dot_product(g1, x1, y1, z1);
#endif
    }

    float t2 = 0.5f - x2 * x2 - y2 * y2 - z2 * z2;
    if (t2 < 0) {
        n2 = 0.0f;
    }
    else {
        t2 *= t2;

#if GF_DEVICE_TARGET_HLSL
        float3 g2;
        get_grad3(gi2, g2);
        n2 = t2 * t2 * dot_product(g2, x2, y2, z2);
#else
        float g2[3];
        get_grad3(gi2, g2);
        n2 = t2 * t2 * dot_product(g2, x2, y2, z2);
#endif
    }

    float t3 = 0.5f - x3 * x3 - y3 * y3 - z3 * z3;
    if (t3 < 0) {
        n3 = 0.0f;
    }
    else {
        t3 *= t3;

#if GF_DEVICE_TARGET_HLSL
        float3 g3;
        get_grad3(gi3, g3);
        n3 = t3 * t3 * dot_product(g3, x3, y3, z3);
#else
        float g3[3];
        get_grad3(gi3, g3);
        n3 = t3 * t3 * dot_product(g3, x3, y3, z3);
#endif
    }

    return 70.0f * (n0 + n1 + n2 + n3);
}


#define F3_CONST 0.333333333f     
#define G3_CONST 0.166666667f     

GF_KERNEL_FUNCTION(AutoShakeKernel,
    ((GF_PTR_READ_ONLY(float4))(inSrc))
    ((GF_PTR(float4))(outDst)),
    ((int)(inSrcPitch))
    ((int)(inDstPitch))
    ((int)(in16f))
    ((unsigned int)(inWidth))
    ((unsigned int)(inHeight))
    ((float)(inMagnitude))
    ((float)(inFrequency))
    ((float)(inEvolution))
    ((float)(inSeed))
    ((float)(inAngle))
    ((float)(inSlack))
    ((float)(inZShake))
    ((int)(inXTiles))
    ((int)(inYTiles))
    ((int)(inMirror))
    ((float)(inCurrentTime))
    ((float)(inDownsampleX))
    ((float)(inDownsampleY))
    ((int)(inNormalMode))
    ((int)(inCompatibilityMode))
    ((float)(inCompatibilityMagnitude))
    ((float)(inCompatibilitySpeed))
    ((float)(inCompatibilityEvolution))
    ((float)(inCompatibilitySeed))
    ((float)(inCompatibilityAngle))
    ((float)(inCompatibilitySlack))
    ((float)(inAccumulatedPhase))  // New parameter for accumulated phase
    ((int)(inHasFrequencyKeyframes)),  // New parameter for keyframe flag
    ((uint2)(inXY)(KERNEL_XY)))
{
    if (inXY.x < inWidth && inXY.y < inHeight)
    {
        if ((inNormalMode == 0 && inCompatibilityMode == 0) ||
            (inNormalMode != 0 && inCompatibilityMode != 0)) {
            float4 pixel = ReadFloat4(inSrc, inXY.y * inSrcPitch + inXY.x, !!in16f);
            WriteFloat4(pixel, outDst, inXY.y * inDstPitch + inXY.x, !!in16f);
            return;
        }

        float angleRad, s, c;
        float evolutionValue;
        float dx, dy, dz;

        if (inNormalMode != 0) {
            angleRad = inAngle * (3.14159265f / 180.0f);
            s = sin(angleRad);
            c = cos(angleRad);

            // Use accumulated phase if available, otherwise use traditional calculation
            if (inHasFrequencyKeyframes != 0) {
                evolutionValue = inEvolution - inAccumulatedPhase;
            }
            else {
                evolutionValue = inEvolution - inFrequency * inCurrentTime;
            }

            dx = simplex_noise(evolutionValue, inSeed * 49235.319798f, 0.0f);
            dy = simplex_noise(evolutionValue + 7468.329f, inSeed * 19337.940385f, 0.0f);
            dz = simplex_noise(evolutionValue + 14192.277f, inSeed * 71401.168533f, 0.0f);

            dx *= inMagnitude * inDownsampleX;
            dy *= inMagnitude * inSlack * inDownsampleY;
            dz *= inZShake;
        }
        else {
            angleRad = inCompatibilityAngle * (3.14159265f / 180.0f);
            s = sin(angleRad);
            c = cos(angleRad);

            evolutionValue = inCompatibilityEvolution +
                (inCurrentTime * inCompatibilitySpeed) -
                inCompatibilitySpeed;

            dx = simplex_noise(inCompatibilitySeed * 54623.245f, 0.0f,
                evolutionValue + inCompatibilitySeed * 49235.319798f);
            dy = simplex_noise(0.0f, inCompatibilitySeed * 8723.5647f,
                evolutionValue + 7468.329f + inCompatibilitySeed * 19337.940385f);

            dx *= inCompatibilityMagnitude * inDownsampleX;
            dy *= inCompatibilityMagnitude * inCompatibilitySlack * inDownsampleY;
            dz = 0.0f;
        }

        dz = -dz;

        float rx = dx * c - dy * s;
        float ry = dx * s + dy * c;

        float srcX = (float)inXY.x - rx;
        float srcY = (float)inXY.y - ry;

        if (dz != 0) {
            float centerX = (float)inWidth / 2.0f;
            float centerY = (float)inHeight / 2.0f;

            float relX = srcX - centerX;
            float relY = srcY - centerY;

            float safe_dz = fmaxf(-900.0f, fminf(900.0f, dz));

            float scale = 1000.0f / (1000.0f - safe_dz);

            scale = fminf(scale, 1.0f);

            srcX = relX / scale + centerX;
            srcY = relY / scale + centerY;
        }

        bool outsideBounds = false;

        if (inXTiles) {
            if (inMirror) {
                float intPart = floor(fabs(srcX / inWidth));
                float fracPart = fabs(srcX / inWidth) - intPart;
                int isOdd = (int)intPart & 1;
                srcX = isOdd ? (1.0f - fracPart) * inWidth : fracPart * inWidth;
            }
            else {
                srcX = fmodf(fmodf(srcX, (float)inWidth) + inWidth, (float)inWidth);
            }
        }
        else {
            if (srcX < 0 || srcX >= inWidth) {
                outsideBounds = true;
            }
        }

        if (inYTiles) {
            if (inMirror) {
                float intPart = floor(fabs(srcY / inHeight));
                float fracPart = fabs(srcY / inHeight) - intPart;
                int isOdd = (int)intPart & 1;
                srcY = isOdd ? (1.0f - fracPart) * inHeight : fracPart * inHeight;
            }
            else {
                srcY = fmodf(fmodf(srcY, (float)inHeight) + inHeight, (float)inHeight);
            }
        }
        else {
            if (srcY < 0 || srcY >= inHeight) {
                outsideBounds = true;
            }
        }

        if (outsideBounds) {
            float4 transparent = { 0.0f, 0.0f, 0.0f, 0.0f };
            WriteFloat4(transparent, outDst, inXY.y * inDstPitch + inXY.x, !!in16f);
            return;
        }

        srcX = fmaxf(0.0f, fminf((float)inWidth - 1.001f, srcX));
        srcY = fmaxf(0.0f, fminf((float)inHeight - 1.001f, srcY));

        int x0 = (int)srcX;
        int y0 = (int)srcY;
        int x1 = min(x0 + 1, (int)inWidth - 1);
        int y1 = min(y0 + 1, (int)inHeight - 1);

        float fx = srcX - x0;
        float fy = srcY - y0;

        float4 p00 = ReadFloat4(inSrc, y0 * inSrcPitch + x0, !!in16f);
        float4 p01 = ReadFloat4(inSrc, y0 * inSrcPitch + x1, !!in16f);
        float4 p10 = ReadFloat4(inSrc, y1 * inSrcPitch + x0, !!in16f);
        float4 p11 = ReadFloat4(inSrc, y1 * inSrcPitch + x1, !!in16f);

        float4 pixel;
        float oneMinusFx = 1.0f - fx;
        float oneMinusFy = 1.0f - fy;

        pixel.x = oneMinusFx * oneMinusFy * p00.x +
            fx * oneMinusFy * p01.x +
            oneMinusFx * fy * p10.x +
            fx * fy * p11.x;

        pixel.y = oneMinusFx * oneMinusFy * p00.y +
            fx * oneMinusFy * p01.y +
            oneMinusFx * fy * p10.y +
            fx * fy * p11.y;

        pixel.z = oneMinusFx * oneMinusFy * p00.z +
            fx * oneMinusFy * p01.z +
            oneMinusFx * fy * p10.z +
            fx * fy * p11.z;

        pixel.w = oneMinusFx * oneMinusFy * p00.w +
            fx * oneMinusFy * p01.w +
            oneMinusFx * fy * p10.w +
            fx * fy * p11.w;

        WriteFloat4(pixel, outDst, inXY.y * inDstPitch + inXY.x, !!in16f);
    }
}

#endif
#if __NVCC__
void AutoShake_CUDA(
    float const* src,
    float* dst,
    unsigned int srcPitch,
    unsigned int dstPitch,
    int is16f,
    unsigned int width,
    unsigned int height,
    float magnitude,
    float frequency,
    float evolution,
    float seed,
    float angle,
    float slack,
    float zshake,
    int x_tiles,
    int y_tiles,
    int mirror,
    float currentTime,
    float downsample_x,
    float downsample_y,
    int normal_mode,
    int compatibility_mode,
    float compatibility_magnitude,
    float compatibility_speed,
    float compatibility_evolution,
    float compatibility_seed,
    float compatibility_angle,
    float compatibility_slack,
    float accumulated_phase,
    int has_frequency_keyframes)
{
    dim3 blockDim(16, 16, 1);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y, 1);

    AutoShakeKernel << < gridDim, blockDim, 0 >> > ((float4 const*)src, (float4*)dst,
        srcPitch, dstPitch, is16f, width, height,
        magnitude, frequency, evolution, seed, angle, slack, zshake,
        x_tiles, y_tiles, mirror, currentTime, downsample_x, downsample_y,
        normal_mode, compatibility_mode, compatibility_magnitude, compatibility_speed,
        compatibility_evolution, compatibility_seed, compatibility_angle, compatibility_slack,
        accumulated_phase, has_frequency_keyframes);

    hipDeviceSynchronize();
}
#endif
#endif

